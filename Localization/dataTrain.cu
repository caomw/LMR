#include "hip/hip_runtime.h"
#include "dataTrain.h"
#include <cudautils.h>
#include "opencv2/features2d.hpp"



DataTrain::DataTrain(const string dataRoot,const string sequenceName){
  pathTodata = dataRoot+"/"+sequenceName+"/";
  siftFrameIDs =NULL;
  numofSiftPerframe =NULL;
  readTraindata(dataRoot,sequenceName,color_list,depth_list,extrinsic,&numofframe,cameraModel);
  this->numofcenters=200;
  this->BOWfeatureTrain= cv::Mat::zeros(numofframe,numofcenters,cv::DataType<float>::type);
  
}
DataTrain::~DataTrain(){
    FreeSiftData(siftDataTrain);
    FreeSiftData(siftDataCenter);
    if (extrinsic!=NULL){
       free(extrinsic);
    }
    if (siftFrameIDs!=NULL){
       free(siftFrameIDs);
    } 
    if (numofSiftPerframe!=NULL){
       free(numofSiftPerframe);
    } 
    BOWfeatureTrain.release();
};
void DataTrain::outputKeyPoint(const string filename){
    FILE *fp = fopen(filename.c_str(),"w");
    fprintf(fp, "ply\n");
    fprintf(fp, "format binary_little_endian 1.0\n");
    fprintf(fp, "element vertex %d\n", totalNumofSift);
    fprintf(fp, "property float x\n");
    fprintf(fp, "property float y\n");
    fprintf(fp, "property float z\n");
    fprintf(fp, "end_header\n");
    for (int i =0;i<totalNumofSift;i++){
        fwrite(siftDataTrain.h_data[i].point3d, sizeof(float), 3, fp);
        
    }
    fclose(fp);
}
void DataTrain::TrainBOW(){
    //Construct BOWKMeansTrainer
    //To store all the descriptors that are extracted from all the images.
    cv::Mat featuresUnclustered;
    int sample =10;
    int i =0;
    while (i<siftDataTrain.numPts){
         cv::Mat descriptor = cv::Mat(1, 128, CV_32FC1, siftDataTrain.h_data[i].data);
         featuresUnclustered.push_back(descriptor);  
         i=i+sample;
    }
    printf("Start clustering with %d of sift\n",featuresUnclustered.size().height);
    //define Term Criteria
    cv::TermCriteria tc(CV_TERMCRIT_ITER,100,0.001);

    //retries number
    int retries=1;
    //necessary flags
    int flags=cv::KMEANS_PP_CENTERS;
    //Create the BoW (or BoF) trainer   
    cv::BOWKMeansTrainer bowTrainer(this->numofcenters,tc,retries,flags);

    //cluster the feature vectors
    cv::Mat dictionary=bowTrainer.cluster(featuresUnclustered);   
    printf("End clustering with %d of sift\n",featuresUnclustered.size().height);
    
    //store the vocabulary
    cv::FileStorage fs(this->pathTodata+"dictionary.yml", cv::FileStorage::WRITE);
    fs << "vocabulary" << dictionary;
    fs.release();

}


void DataTrain::preComputeBOW(){
    if (!exists(this->pathTodata+"dictionary.yml")){
      this->TrainBOW();
    }
    cv::Mat dictionary;
    cv::FileStorage fs(this->pathTodata+"dictionary.yml", cv::FileStorage::READ);
    fs["vocabulary"] >> dictionary;
    fs.release();  
    InitSiftData(siftDataCenter, numofcenters, true, true);
    siftDataCenter.numPts = numofcenters;
    for (int c_id =0; c_id < numofcenters; c_id++)
    {
      //float* ptr = dictionary.ptr<float>(c_id);
      //for (int i =0;i<128;i++){
      memcpy(siftDataCenter.h_data[c_id].data,dictionary.ptr<float>(c_id),sizeof(float)*128);
      //}
      
    }
    safeCall(hipMemcpy(siftDataCenter.d_data, siftDataCenter.h_data, 
             sizeof(SiftPoint)*siftDataCenter.numPts, hipMemcpyHostToDevice));
  

    if (0){//(exists(this->pathTodata+"BOWTrain")){
       printf("loading BOWTrain ...\n");
       FILE * pFile = fopen((this->pathTodata+"BOWTrain").c_str(),"wb");
       fread(BOWfeatureTrain.data, sizeof(float),numofframe*numofcenters,pFile);
       fclose(pFile);
    }
    else{
      printf("computing BOWTrain ...\n");
      MatchSiftData(this->siftDataTrain,this->siftDataCenter);
      for (int i =0;i<siftDataTrain.numPts;i++){
          BOWfeatureTrain.at<float>(siftFrameIDs[i],this->siftDataTrain.h_data[i].match) =
          BOWfeatureTrain.at<float>(siftFrameIDs[i],this->siftDataTrain.h_data[i].match)+1;
      }
      cv::Mat BOWfeatureTrain_summed;
      cv::reduce(BOWfeatureTrain, BOWfeatureTrain_summed, 1, CV_REDUCE_SUM, cv::DataType<float>::type);
/*
FILE * ss = fopen("siftFrameIDs.bin","wb");
fwrite(siftFrameIDs, sizeof(int),totalNumofSift,ss);
fclose(ss);      
writeMatToFile(BOWfeatureTrain, "BOWfeatureTrain.txt");
writeMatToFile(BOWfeatureTrain_summed, "BOWfeatureTrain_summed.txt");
*/
//cout<<BOWfeatureTrain.row(0)<<endl;
      //cout <<"BOWfeatureTrain_summed"<<BOWfeatureTrain_summed.size().height<<","<<BOWfeatureTrain_summed.size().width<<endl;
      for (int i =0;i<BOWfeatureTrain.size().height;i++){
          for (int j = 0;j<BOWfeatureTrain.size().width;j++){
            BOWfeatureTrain.at<float>(i,j) = BOWfeatureTrain.at<float>(i,j)/(BOWfeatureTrain_summed.at<float>(i)+1);
          }
          
      }
      FILE * pFile = fopen((this->pathTodata+"BOWTrain").c_str(),"wb");
      fwrite(BOWfeatureTrain.data, sizeof(float),numofframe*numofcenters,pFile);
      fclose(pFile);
    }
}
cv::Mat DataTrain::findNearFrameBOW(const cv::Mat &BOWTest){
        cv::Mat distance = cv::Mat(this->numofframe,1,cv::DataType<float>::type);
        for (int i =0;i<this->numofframe;i++){
            cv::Mat subtrans = (this->BOWfeatureTrain.row(i) - BOWTest);
            float sum = 0;
            for (int j = 0;j<this->numofcenters;j++){
               sum += subtrans.at<float>(j) *subtrans.at<float>(j);
            }
           distance.at<float>(i) = sum;
        }
        
       cv::Mat dst;
       cv::sortIdx(distance, dst, CV_SORT_EVERY_COLUMN + CV_SORT_ASCENDING);
       return dst;
}

void DataTrain::outputPly(const string filename, int numofsample=10000){
  int numofPtsperImage = floor(numofsample/numofframe);
  numofsample = numofPtsperImage*numofframe;
  for (int frame_id =0; frame_id < this->numofframe; frame_id++){
    cv::Mat limg = cv::imread(this->color_list[frame_id]);
    cv::Mat ldepth = GetDepthData(this->depth_list[frame_id]);
    cv::Mat pointCloud_l = depth2XYZcamera(this->cameraModel,ldepth,1);
    pointCloud_l = transformPointCloud(pointCloud_l,&(this->extrinsic[12*frame_id]));
    char buffer [50];
    sprintf (buffer, "%s%d.ply", filename.c_str(), frame_id);
    WritePlyFile(buffer, pointCloud_l, limg);
  }
}

void DataTrain::preComputeSift(const string filename)
{
  // for each Train image compute and save sift
  this->totalNumofSift = 0;
  FILE * pFile = fopen((this->pathTodata+filename).c_str(),"wb");
  for (int frame_id =0; frame_id < this->numofframe; frame_id++){
    // get sift

    cv::Mat limg = cv::imread(this->color_list[frame_id], 0);
    SiftData siftData1 = computeSift(limg);
    cv::Mat ldepth = GetDepthData(this->depth_list[frame_id]);
    // get points and rotate the point to world cordinate
    cv::Mat pointCloud_l = depth2XYZcamera(this->cameraModel,ldepth,1);
    pointCloud_l = transformPointCloud(pointCloud_l,&(this->extrinsic[12*frame_id]));
    int numofvalid = getSift3dPoints(siftData1,pointCloud_l,ldepth.size().width);
//cout<<this->color_list[frame_id]<<endl;
    cout<<"numofvalid:"<<numofvalid<<endl;

    SiftPoint *sift1 = siftData1.h_data;
    for (int i = 0;i<siftData1.numPts;i++){
      if ( sift1[i].valid >0){
        fwrite(&sift1[i].xpos,sizeof(float),1,pFile);
        fwrite(&sift1[i].ypos,sizeof(float),1,pFile);
        fwrite(&frame_id,sizeof(int),1,pFile);
        fwrite(sift1[i].data,sizeof(float),128,pFile);
        fwrite(sift1[i].point3d,sizeof(float),3,pFile);
        this->totalNumofSift++;
      }
    }
    FreeSiftData(siftData1);
  }

  fclose(pFile);
  FILE * pFile2 = fopen((this->pathTodata+filename+"Num").c_str(),"wb");
  fwrite(&this->totalNumofSift,sizeof(int),1,pFile2);
  fclose(pFile2);
  printf("totalNumofSift: %d\n",this->totalNumofSift);
}

void DataTrain::loadComputedSift(const string filename)
{
  FILE * pFile2 = fopen((this->pathTodata+filename+"Num").c_str(),"rb");
  fread(&totalNumofSift,sizeof(int),1,pFile2);
  
  InitSiftData(siftDataTrain, totalNumofSift, true, true);
  siftDataTrain.maxPts = siftDataTrain.numPts = totalNumofSift;  
  this->siftFrameIDs = (int *)malloc(sizeof(int)*totalNumofSift);

  this->numofSiftPerframe = (int *)malloc(sizeof(int)*this->numofframe);
  for (int i = 1;i<this->numofframe;i++){
      numofSiftPerframe[i]=0;
  }
  // load all sift of 
  FILE * pFile = fopen((this->pathTodata+filename).c_str(),"rb");
  for (int sift_id =0; sift_id < this->totalNumofSift; sift_id++)
  {
     fread(&siftDataTrain.h_data[sift_id].xpos,sizeof(float),1,pFile);
     fread(&siftDataTrain.h_data[sift_id].ypos,sizeof(float),1,pFile);
     fread(&siftFrameIDs[sift_id], sizeof(int), 1, pFile);
     fread(siftDataTrain.h_data[sift_id].data,sizeof(float),128,pFile);
     fread(siftDataTrain.h_data[sift_id].point3d,sizeof(float),3,pFile);
     numofSiftPerframe[siftFrameIDs[sift_id]] = numofSiftPerframe[siftFrameIDs[sift_id]]+1;
  }
  
  

  safeCall(hipMemcpy(siftDataTrain.d_data, siftDataTrain.h_data, sizeof(SiftPoint)*siftDataTrain.numPts, hipMemcpyHostToDevice));
  
  printf("totalNumofSift: %d size:%lu\n ",siftDataTrain.numPts,sizeof(SiftPoint)*siftDataTrain.numPts);
  return;
}


SiftData computeSift(cv::Mat inputImage){
  //InitCuda();
  inputImage.convertTo(inputImage, CV_32FC1);
  unsigned int w = inputImage.cols;
  unsigned int h = inputImage.rows;
  //std::cout << "Image size = (" << w << "," << h << ")" << std::endl;
  cv::GaussianBlur(inputImage, inputImage, cv::Size(5,5), 1.0);
  
  //std::cout << "Initializing data..." << std::endl;
  CudaImage img1;
  img1.Allocate(w, h, iAlignUp(w, 128), false, NULL, (float*)inputImage.data);
  //(int w, int h, int p, bool host, float *devmem, float *hostmem) 
  img1.Download();
  SiftData siftData1;
  InitSiftData(siftData1, 2048, true, true);
  float initBlur = 0.0f;
  float thresh = 4.0f; 
  double timesift1 = ExtractSift(siftData1, img1, 5, initBlur, thresh, 0.0f);
  std::cout << "Extract sift time: " <<  timesift1 <<"ms"<< std::endl;
  std::cout << "Number of original features: " <<  siftData1.numPts << std::endl;
  return siftData1;
}


void readTraindata(const string dataRoot,const string sequenceName,
                   vector<string> &color_list,vector<string>& depth_list,
                   float* &extrinsic,int* numofframe, cameraModel &cam_K)
{
    // color
    string listfile_color = dataRoot+sequenceName+"colorTrain.txt";
    string line;
    ifstream file_color (listfile_color);
    if (file_color.is_open()){
      getline(file_color,line);
      *numofframe = atoi( line.c_str() );
      while(getline(file_color,line)){
           color_list.push_back(line);
      }
      file_color.close();
    }
    else cout << "Unable to open file: "<< listfile_color;
    
    
    //depth
    string listfile_depth = dataRoot+sequenceName+"depthTrain.txt";
    ifstream  myfile(listfile_depth);
    if (myfile.is_open()){
      getline(myfile,line);
      while(getline(myfile,line)){
            depth_list.push_back(line);
      }
      myfile.close();
    }
    else cout << "Unable to open file: "<< listfile_color;

    // extrinsic
    string extrinsic_file = dataRoot+sequenceName+"extrinsics.txt";
    fstream ex_myfile(extrinsic_file);
    if (ex_myfile.is_open()){
      extrinsic = (float*) malloc (sizeof(float)*numofframe[0]*12);
      for (int i=0;i<12*numofframe[0];i++){
           ex_myfile>>extrinsic[i];
      }
      ex_myfile.close();
    }

    string cam_file = dataRoot+sequenceName+"/intrinsics.txt";
    ifstream  cam_myfile(cam_file);
    float tmp;
    if (cam_myfile.is_open()){
        cam_myfile>>cam_K.fx;
        cam_myfile>>tmp;
        cam_myfile>>cam_K.cx;
        cam_myfile>>tmp;
        cam_myfile>>cam_K.fy;
        cam_myfile>>cam_K.cy;
        cam_myfile.close();
        //cout << cam_K.fx <<","<< cam_K.fy <<","<<cam_K.cx<<","<<cam_K.cy<<endl;
    }
    else cout << "Unable to open file: "<< cam_file;
    return;
}

int getSift3dPoints(SiftData siftData1,const cv::Mat pointCloud_l,const int imgw)
{
    int numTomatchedsift =0;
    SiftPoint *sift1 = siftData1.h_data;
    for (int i = 0;i<siftData1.numPts;i++){
        int ind  = ((int)sift1[i].xpos+(int)sift1[i].ypos*imgw);
        const float *ptr = (float*)pointCloud_l.ptr(ind);  

        if (ptr[3]>0.0001) {  
            numTomatchedsift++;
            sift1[i].valid =1;
            sift1[i].point3d[0] = ptr[0];
            sift1[i].point3d[1] = ptr[1];
            sift1[i].point3d[2] = ptr[2];
        }
        else{
            sift1[i].valid =-1;
        }
    } 
    return numTomatchedsift;
}